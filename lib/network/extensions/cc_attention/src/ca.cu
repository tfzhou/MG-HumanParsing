#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>

#include "common.h"
#include "ca.h"


__global__ void ca_forward_kernel(const float *t, const float *f, float *weight, int num, int chn, int height, int width) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int sp = height * width;
  int len = height + width - 1;
  int z = blockIdx.z;

  if (x < width && y < height && z < height+width-1) {
    for (int batch = 0; batch < num; ++batch) {
      for (int plane = 0; plane < chn; ++plane) {
        float _t = t[(batch * chn + plane) * sp + y*width + x];

        if (z < width) {
          int i = z;
          float _f = f[(batch * chn + plane) * sp + y*width + i];
          weight[(batch * len + i) * sp + y*width + x] += _t*_f;
        } else {
          int i = z - width;
          int j = i<y ? i : i+1;

          float _f = f[(batch * chn + plane) * sp + j*width + x];
          weight[(batch * len + width + i) * sp + y*width + x] += _t*_f;
        }
      }
    }
  }
}

__global__ void ca_backward_kernel_t(const float *dw, const float *t, const float *f, float *dt,
                                int num, int chn, int height, int width) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int sp = height * width;
  int len = height + width - 1;
  int plane = blockIdx.z;

  if (x < width && y < height && plane < chn) {
    for (int batch = 0; batch < num; ++batch) {
        
        for (int i = 0; i < width; ++i) {
          float _dw = dw[(batch * len + i) * sp + y*width + x];
          float _f = f[(batch * chn + plane) * sp + y*width + i];
          dt[(batch * chn + plane) * sp + y*width + x] += _dw * _f;
        }
        for (int i = 0; i < height; ++i)  {
          if (i == y) continue;
          int j = i<y ? i : i-1;

          float _dw = dw[(batch * len + width + j) * sp + y*width + x];
          float _f = f[(batch * chn + plane) * sp + i*width + x];
          dt[(batch * chn + plane) * sp + y*width + x] += _dw * _f;
        }
    }

  }
}

__global__ void ca_backward_kernel_f(const float *dw, const float *t, const float *f, float *df, 
                                int num, int chn, int height, int width) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int sp = height * width;
  int len = height + width - 1;
  int plane = blockIdx.z;

  if (x < width && y < height && plane < chn) {
    
    for (int batch = 0; batch < num; ++batch) {
      
      for (int i = 0; i < width; ++i) {
        float _dw = dw[(batch * len + x) * sp + y*width + i];
        float _t = t[(batch * chn + plane) * sp + y*width + i];
        df[(batch * chn + plane) * sp + y*width + x] += _dw * _t;
      }
      for (int i = 0; i < height; ++i) {
        if (i == y) continue;
        int j = i>y ? y : y-1;

        float _dw = dw[(batch * len + width + j) * sp + i*width + x];
        float _t = t[(batch * chn + plane) * sp + i*width + x];
        df[(batch * chn + plane) * sp + y*width + x] += _dw * _t;
      }
    }

  }
}


__global__ void ca_map_forward_kernel(const float *weight, const float *g, float *out, int num, int chn, int height, int width) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int sp = height * width;
  int len = height + width - 1;
  int plane = blockIdx.z;

  if (x < width && y < height && plane < chn) {
    for (int batch = 0; batch < num; ++batch) {

      for (int i = 0; i < width; ++i) {
        float _g = g[(batch * chn + plane) * sp + y*width + i];
        float _w = weight[(batch * len + i) * sp + y*width + x];
        out[(batch * chn + plane) * sp + y*width + x] += _g * _w;
      }
      for (int i = 0; i < height; ++i) {
        if (i == y) continue;

        int j = i<y ? i : i-1;

        float _g = g[(batch * chn + plane) * sp + i*width + x];
        float _w = weight[(batch * len + width + j) * sp + y*width + x];
        out[(batch * chn + plane) * sp + y*width + x] += _g * _w;
      }
    }
  }

}

__global__ void ca_map_backward_kernel_w(const float *dout, const float *weight, const float *g, float *dw,
                                int num, int chn, int height, int width) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int sp = height * width;
  int len = height + width - 1;
  int z = blockIdx.z;

  if (x < width && y < height && z < height+width-1) {

    for (int batch = 0; batch < num; ++batch) {
      for (int plane = 0; plane < chn; ++plane) {
        float _dout = dout[(batch * chn + plane) * sp + y*width + x];

        if (z < width) {
          int i = z;
          float _g = g[(batch * chn + plane) * sp + y*width + i];
          dw[(batch * len + i) * sp + y*width + x] += _dout * _g;
        } else {
          int i = z - width;
          int j = i<y ? i : i+1;

          float _g = g[(batch * chn + plane) * sp + j*width + x];
          dw[(batch * len + width + i) * sp + y*width + x] += _dout * _g;
        }
      }
    }
  }
}

__global__ void ca_map_backward_kernel_g(const float *dout, const float *weight, const float *g, float *dg, 
                                int num, int chn, int height, int width) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int sp = height * width;
  int len = height + width - 1;
  int plane = blockIdx.z;

  if (x < width && y < height && plane < chn) {

    for (int batch = 0; batch < num; ++batch) {
      for (int i = 0; i < width; ++i) {
        float _dout = dout[(batch * chn + plane) * sp + y*width + i];
        float _w = weight[(batch * len + x) * sp + y*width + i];
        dg[(batch * chn + plane) * sp + y*width + x] += _dout * _w;
      }

      for (int i = 0; i < height; ++i) {
        if (i == y) continue;
        int j = i>y ? y : y-1;

        float _dout = dout[(batch * chn + plane) * sp + i*width + x];
        float _w = weight[(batch * len + width + j) * sp + i*width + x];
        dg[(batch * chn + plane) * sp + y*width + x] += _dout * _w;
      }
    }
  }
}

/*
 * Implementations
 */
extern "C" int _ca_forward_cuda(int N, int C, int H, int W, const float *t, 
                                const float *f, float *weight, hipStream_t stream) {
  // Run kernel
  dim3 threads(32, 32);
  int d1 = (W+threads.x-1)/threads.x;
  int d2 = (H+threads.y-1)/threads.y;
  int d3 = H+W;
  dim3 blocks(d1, d2, d3);
  ca_forward_kernel<<<blocks, threads, 0, stream>>>(t, f, weight, N, C, H, W);

  // Check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return 0;
  else
    return 1;
}


extern "C" int _ca_backward_cuda(int N, int C, int H, int W, const float *dw, const float *t, const float *f, float *dt, float *df, hipStream_t stream) {
  // Run kernel
  dim3 threads(32, 32);
  int d1 = (W+threads.x-1)/threads.x;
  int d2 = (H+threads.y-1)/threads.y;
  int d3 = C;
  dim3 blocks(d1, d2, d3);
  // printf("%f\n", dw[0]);
  ca_backward_kernel_t<<<blocks, threads, 0, stream>>>(dw, t, f, dt, N, C, H, W);
  ca_backward_kernel_f<<<blocks, threads, 0, stream>>>(dw, t, f, df, N, C, H, W);

  // Check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return 0;
  else
    return 1;
}


extern "C" int _ca_map_forward_cuda(int N, int C, int H, int W, const float *weight, const float *g, float *out, hipStream_t stream) {
  // Run kernel
  dim3 threads(32, 32);
  dim3 blocks((W+threads.x-1)/threads.x, (H+threads.y-1)/threads.y, C);
  ca_map_forward_kernel<<<blocks, threads, 0, stream>>>(weight, g, out, N, C, H, W);

  // Check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return 0;
  else
    return 1;
}

extern "C" int _ca_map_backward_cuda(int N, int C, int H, int W, const float *dout, const float *weight, const float *g, float *dw, float *dg, hipStream_t stream) {
  // Run kernel
  dim3 threads(32, 32);
  int d1 = (W+threads.x-1)/threads.x;
  int d2 = (H+threads.y-1)/threads.y;
  int d3 = H+W;
  dim3 blocks(d1, d2, d3);
  ca_map_backward_kernel_w<<<blocks, threads, 0, stream>>>(dout, weight, g, dw, N, C, H, W);

  d3 = C;
  blocks = dim3(d1, d2, d3);
  ca_map_backward_kernel_g<<<blocks, threads, 0, stream>>>(dout, weight, g, dg, N, C, H, W);

  // Check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return 0;
  else
    return 1;
}
